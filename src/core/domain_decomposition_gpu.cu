#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <cstdlib>
#include <iostream>
#include <ctime>

#include "domain_decomposition_gpu.hpp"
#include "cuda_utils.hpp"

/* Forward declarations of gpu kernels. */

__global__ static void sortParticlesGenerateCellist(unsigned int n_part, const float3 *xyz, float3 *xyz_sorted, unsigned int *hashes, unsigned int *indexes, uint2 *cells);

__global__ static void hashAtoms(unsigned int n_part, float3 hi, const float3 *xyz, unsigned int *index, unsigned int *hashes, uint3 n_cells);

/* Class Implementation */

DomainDecompositionGpu::DomainDecompositionGpu(float3 _box, unsigned int _n_part, uint3 _n_cells) {

  n_part = _n_part;
  box = _box;
  n_cells = _n_cells;
  hi.x = n_cells.x/box.x;
  hi.y = n_cells.y/box.y;
  hi.z = n_cells.z/box.z;
  total_cells = n_cells.x*n_cells.y*n_cells.z;

  init_device_memory(true, true);
}

DomainDecompositionGpu::~DomainDecompositionGpu() {
  free_device_memory(true, true);
}

void DomainDecompositionGpu::free_device_memory(bool particles, bool dd) {
  if(particles) {
    cuda_safe_mem(hipFree(indexes));
    cuda_safe_mem(hipFree(hashes));
    cuda_safe_mem(hipFree(xyz_sorted));
  }
  if(dd) {
    cuda_safe_mem(hipFree(cells));
  }
}

void DomainDecompositionGpu::init_device_memory(bool particles, bool dd) {
  if(particles) {
    cuda_safe_mem(hipMalloc((void **)&(indexes), n_part*sizeof(unsigned int)));
    cuda_safe_mem(hipMalloc((void **)&(hashes), n_part*sizeof(unsigned int)));
    cuda_safe_mem(hipMalloc((void **)&(xyz_sorted), n_part*sizeof(float3)));
  }
  if(dd) {
    cuda_safe_mem(hipMalloc((void **)&(cells), total_cells*sizeof(uint2)));
  }
}

void DomainDecompositionGpu::build(float3 *xyz) {
  dim3 block(256,1,1), grid(1,1,1);
  
  grid.x = max(1, (n_part + block.x - 1) / block.x);

  #ifdef CUDA_DD_DEBUG
  printf("build_dd() grid.x = %d, block.x = %d\n", grid.x, block.x);
  #endif

  cuda_safe_mem(hipMemset(cells, CELL_EMPTY, total_cells*sizeof(uint2)));

  hashAtoms<<<grid, block>>>(n_part, hi, xyz, indexes, hashes, n_cells);

  thrust::sort_by_key(thrust::device_ptr<unsigned int>(hashes),
		      thrust::device_ptr<unsigned int>(hashes + n_part),
		      thrust::device_ptr<unsigned int>(indexes));

  unsigned int smemsize = sizeof(unsigned int)*(block.x + 1);

  sortParticlesGenerateCellist<<<grid,block,smemsize>>>(n_part, xyz, xyz_sorted, hashes, indexes, cells);
}

void DomainDecompositionGpu::set_n_part(unsigned int _n_part) {
  n_part = _n_part;
  free_device_memory(true, false);
  init_device_memory(true, false);
}

/* GPU Kernels */

__global__ static void sortParticlesGenerateCellist(unsigned int n_part, const float3 *xyz, float3 *xyz_sorted, unsigned int *hashes, unsigned int *indexes, uint2 *cells) {
  unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
  unsigned int hash;

  extern __shared__ unsigned int hash_s[];

  if(id < n_part) {
    hash = hashes[id];
    hash_s[threadIdx.x+1] = hash;
    if( (id > 0) && (threadIdx.x == 0))
      hash_s[0] = hashes[id-1];
  }

  __syncthreads();
  
  if(id < n_part) {
    if( (id == 0) || hash != hash_s[threadIdx.x] ) {
      cells[hash].x = id;
      if(id > 0)
	cells[hash_s[threadIdx.x]].y = id;
    }
    if( id == n_part - 1)
      cells[hash].y = id + 1;

    unsigned int sorted_id;
    sorted_id = indexes[id];
    xyz_sorted[id] = xyz[sorted_id];
  }
}

__global__ static void hashAtoms(unsigned int n_part, float3 hi, const float3 *xyz, unsigned int *index, unsigned int *hashes, uint3 n_cells) {
  unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;

  if(id >= n_part)
    return;

  int3 cell;
  float3 part = xyz[id];  

  cell.x = ((int)(part.x * hi.x + 0.5)) % n_cells.x;
  cell.y = ((int)(part.y * hi.y + 0.5)) % n_cells.y;
  cell.z = ((int)(part.z * hi.z + 0.5)) % n_cells.z;

  hashes[id] =  cell.x * n_cells.y * n_cells.z +  cell.y * n_cells.z + cell.z;
  index[id] = id;
}

__device__ static inline float dist2(float3 a, float3 b) {
  return ( (a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y) + (a.z - b.z) * (a.z - b.z));
}

__global__ static void nearestNeighbors(unsigned int n_part, uint3 n_cells, const uint2 *cells, const float3 *xyz, unsigned int *neighbors, int range, float rcut2) {
  unsigned int xindex = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int yindex = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int zindex = blockDim.z * blockIdx.z + threadIdx.z;
  int n,m,l;
  unsigned int cellhash = n_cells.y * n_cells.z * xindex + n_cells.z * yindex + zindex;
  int hash;
  unsigned int m_neighbors = 0;

  if( (xindex >= n_cells.x) || (yindex >= n_cells.y) || (zindex >= n_cells.z))
    return;

  for(unsigned int id = cells[cellhash].x; id != cells[cellhash].y; id++) {
    m_neighbors = 0;
    float3 parti = xyz[id];
    for(int i = -range; i <= range; i++) {
      n = xindex + i;
      if((n < 0) || (n >= n_cells.x))
	continue;
      for(int j = -range; j <= range; j++) {
	m = yindex + j;
	if((m < 0) || (m >= n_cells.y))
	  continue;
	for(int k = -range; k <= range; k++) {
	  l = zindex + k;
	  if((l < 0) || (l >= n_cells.z))
	    continue;

	  hash = n_cells.y * n_cells.z * n + n_cells.z * m + l;
	  if( (cells[hash].x == CELL_EMPTY))
	    continue;
	  for(unsigned int jd = cells[hash].x; jd != cells[hash].y; jd++) {
	    if(dist2(parti, xyz[jd]) <= rcut2) {
	      m_neighbors++;
	    }
	  }
	}
      }
    }
    neighbors[id] = m_neighbors;
  }
}

__global__ static void nearestNeighbors_n2(unsigned int n_part, const float3 *xyz, unsigned int *neighbors, float rcut2) {
  unsigned int id = blockIdx.x;
  unsigned int m_neighbors = 0;

  if(id >= n_part)
    return;

  float3 parti = xyz[id]; 

  for(unsigned int i = 0; i < n_part; i++) {
    if(dist2(parti, xyz[i]) <= rcut2) {
      m_neighbors++;
    }    
  }
  neighbors[id] = m_neighbors;
}


// void print_dd(dd_t *dd) {
//   puts("print_dd()");
//   uint2 *cells_h;
//   float3 *xyz_h;
//   unsigned int count = 0;

//   cells_h = (uint2 *)malloc(dd->total_cells*sizeof(uint2));
//   xyz_h = (float3 *)malloc(dd->n_part*sizeof(float3));

//   hipMemcpy(cells_h, dd->cells, dd->total_cells*sizeof(uint2), hipMemcpyDeviceToHost);
//   hipMemcpy(xyz_h, dd->xyz_sorted, dd->n_part*sizeof(float3), hipMemcpyDeviceToHost);  

//   printf("dd { n_part = %d, n_cells = ( %d %d %d ), hi = ( %f %f %f ) }\n", dd->n_part, dd->n_cells.x, dd->n_cells.y, dd->n_cells.z
// 	 , dd->hi.x, dd->hi.y, dd->hi.z);

//   bool *part_map = (bool *)malloc(dd->n_part*sizeof(bool));
//   for(int i = 0; i < dd->n_part; i++)
//     part_map[i] = false;
  
//   for(int i = 0; i < dd->total_cells; i++) {
//     if(cells_h[i].x == CELL_EMPTY)
//       continue;
//     unsigned int x,y,z;
//     z = i % dd->n_cells.z;
//     y = ((i - z)/dd->n_cells.z) % dd->n_cells.y;
//     x = (i - z - dd->n_cells.z*y) / (dd->n_cells.z*dd->n_cells.y);
//     printf("cell %d, pos (%d %d %d), center (%f %f %f)\n", i, x, y, z, x/dd->hi.x, y/dd->hi.y, z/dd->hi.z);
//     for(unsigned int it = cells_h[i].x; it != cells_h[i].y; it++) {
//       part_map[it] = true;
//       printf("\t%d: pos = (%f %f %f %f), dist = (%f %f %f), dist/h = (%f %f %f)\n", it,xyz_h[it].x,xyz_h[it].y,xyz_h[it].z,0,
// 	     xyz_h[it].x-x/dd->hi.x, xyz_h[it].y-y/dd->hi.y, xyz_h[it].z-z/dd->hi.z,
// 	     dd->hi.x*(xyz_h[it].x-x/dd->hi.x), dd->hi.y*(xyz_h[it].y-y/dd->hi.y), dd->hi.z*(xyz_h[it].z-z/dd->hi.z) );
//       count++;
//     }
//   }
//   printf("%d particles in dd.\n", count);
//   for(int i = 0; i < dd->n_part; i++)
//     if(!part_map[i])
//       printf("particle %d is missing.\n",i);

// }

