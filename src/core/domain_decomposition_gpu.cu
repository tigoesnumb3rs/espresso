#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <cstdlib>
#include <iostream>
#include <ctime>

#include "domain_decomposition_gpu.hpp"
#include "domain_decomposition_gpu_test.hpp"
#include "cuda_utils.hpp"

#ifdef GPU_DD_DEBUG
void print_dd(DomainDecompositionGpu &dd);
#endif

/* Forward declarations of gpu kernels. */

__global__ static void sortParticlesGenerateCellist(unsigned int n_part, const float3 *xyz, float3 *xyz_sorted, unsigned int *hashes, unsigned int *indexes, uint2 *cells);

__global__ static void hashAtoms(unsigned int n_part, float3 hi, const float3 *xyz, unsigned int *index, unsigned int *hashes, uint3 n_cells);

template<typename T>
__global__ static void unsortArrays(unsigned int *indexes, T *src, T *dst, unsigned int n);

/* Class Implementation */

DomainDecompositionGpu::DomainDecompositionGpu(float3 _box, unsigned int _n_part, uint3 _n_cells) : n_part(_n_part), box(_box) {

  n_cells = _n_cells;
  hi.x = n_cells.x/box.x;
  hi.y = n_cells.y/box.y;
  hi.z = n_cells.z/box.z;
  total_cells = n_cells.x*n_cells.y*n_cells.z;

  init_device_memory(true, true);
}

DomainDecompositionGpu::DomainDecompositionGpu(float3 _box, unsigned int _n_part, float cutoff) : n_part(_n_part), box(_box) {
  /* Need to take the floor to be on the safe side */
  n_cells.x = floor(box.x / cutoff);
  n_cells.y = floor(box.y / cutoff);
  n_cells.z = floor(box.z / cutoff);

  hi.x = n_cells.x/box.x;
  hi.y = n_cells.y/box.y;
  hi.z = n_cells.z/box.z;
  total_cells = n_cells.x*n_cells.y*n_cells.z;

  init_device_memory(true, true);
}

DomainDecompositionGpu::~DomainDecompositionGpu() {
  free_device_memory(true, true);
}

void DomainDecompositionGpu::free_device_memory(bool particles, bool dd) {
  if(particles) {
    cuda_safe_mem(hipFree(indexes));
    cuda_safe_mem(hipFree(hashes));
    cuda_safe_mem(hipFree(xyz_sorted));
  }
  if(dd) {
    cuda_safe_mem(hipFree(cells));
  }
}

void DomainDecompositionGpu::init_device_memory(bool particles, bool dd) {
  if(particles) {
    cuda_safe_mem(hipMalloc((void **)&(indexes), n_part*sizeof(unsigned int)));
    cuda_safe_mem(hipMalloc((void **)&(hashes), n_part*sizeof(unsigned int)));
    cuda_safe_mem(hipMalloc((void **)&(xyz_sorted), n_part*sizeof(float3)));
  }
  if(dd) {
    cuda_safe_mem(hipMalloc((void **)&(cells), total_cells*sizeof(uint2)));
  }
}

void DomainDecompositionGpu::build(float3 *xyz) {
  dim3 block(1,1,1), grid(1,1,1);

  if(n_part < 128) {
    block.x = n_part;
    grid.x = 1;    
  } else {
    block.x = 128;
    grid.x = max(1, (n_part + block.x - 1) / block.x);
  }

  cuda_safe_mem(hipMemset(cells, CELL_EMPTY, total_cells*sizeof(uint2)));

  KERNELCALL(hashAtoms, grid, block, (n_part, hi, xyz, indexes, hashes, n_cells));

  thrust::sort_by_key(thrust::device_ptr<unsigned int>(hashes),
  		      thrust::device_ptr<unsigned int>(hashes + n_part),
  		      thrust::device_ptr<unsigned int>(indexes));

  unsigned int smemsize = sizeof(unsigned int)*(block.x + 1);

  KERNELCALL_shared(sortParticlesGenerateCellist, grid, block, smemsize, (n_part, xyz, xyz_sorted, hashes, indexes, cells));
}

void DomainDecompositionGpu::set_n_part(unsigned int _n_part) {
  n_part = _n_part;  
  free_device_memory(true, false);
  init_device_memory(true, false);
}

template<typename T>
void DomainDecompositionGpu::unsort(T *src, T* dst) {
  dim3 block(1,1,1), grid(1,1,1);

  if(n_part < 128) {
    block.x = n_part;
    grid.x = 1;    
  } else {
    block.x = 128;
    grid.x = max(1, (n_part + block.x - 1) / block.x);
  }

  KERNELCALL(unsortArrays<T>, grid, block, (indexes, src, dst, n_part));
}

/* GPU Kernels */

template<typename T>
__global__ static void unsortArrays(unsigned int *indexes, T *src, T *dst, unsigned int n) {
  unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;

  if(id >= n)
    return;

  dst[indexes[id]] = src[id];
}

__global__ static void sortParticlesGenerateCellist(unsigned int n_part, const float3 *xyz, float3 *xyz_sorted, unsigned int *hashes, unsigned int *indexes, uint2 *cells) {
  unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
  unsigned int hash;

  extern __shared__ unsigned int hash_s[];

  if(id < n_part) {
    hash = hashes[id];
    hash_s[threadIdx.x+1] = hash;
    if( (id > 0) && (threadIdx.x == 0))
      hash_s[0] = hashes[id-1];
  }

  __syncthreads();
  
  if(id < n_part) {
    if( (id == 0) || hash != hash_s[threadIdx.x] ) {
      cells[hash].x = id;
      if(id > 0)
	cells[hash_s[threadIdx.x]].y = id;
    }
    if( id == n_part - 1)
      cells[hash].y = id + 1;

    unsigned int sorted_id;
    sorted_id = indexes[id];
    xyz_sorted[id] = xyz[sorted_id];
  }
}

__global__ static void hashAtoms(unsigned int n_part, float3 hi, const float3 *xyz, unsigned int *index, unsigned int *hashes, uint3 n_cells) {
  unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;

  if(id >= n_part)
    return;

  int3 cell;
  float3 part = xyz[id];  

  cell.x = ((int)(part.x * hi.x)) % n_cells.x;
  cell.y = ((int)(part.y * hi.y)) % n_cells.y;
  cell.z = ((int)(part.z * hi.z)) % n_cells.z;

  hashes[id] =  cell.x * n_cells.y * n_cells.z +  cell.y * n_cells.z + cell.z;
  index[id] = id;
}

  /** Unit test for the GPU Domain Decomposition
      ------------------------------------------
      The testing strategy is as follows:
      First the memory management is testet by changing the number of particles and
      the cutoff/number of cells and see if this is handled correctly.
      Then the function is testet by calculation the neighbor count of a know particle
      configuration. This is also calculated via an nsquare algorithm to check its implementation. 
      Then the neighbor count of a random system is compared between the calculation with domain decomposition and the nquared algorithm.
  **/

  /* Functions for unit testing */
static bool test_memory_management();
static bool test_neighbor_count();
__global__ static void nearestNeighbors(unsigned int n_part, uint3 n_cells, const uint2 *cells, const float3 *xyz, unsigned int *neighbors, float rcut2);
__global__ static void nearestNeighbors_n2(unsigned int n_part, const float3 *xyz, unsigned int *neighbors, float rcut2);

bool domain_decomposition_gpu_unit_test() {
  bool result = true;

  result &= test_memory_management();
  result &= test_neighbor_count();

  return result;
}

__device__ static inline float dist2(float3 a, float3 b) {
  return ( (a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y) + (a.z - b.z) * (a.z - b.z));
}

static bool test_memory_management() {
  return false;
}

bool test_decomposition(DomainDecompositionGpu &dd) {
  uint2 *cells_h;
  float3 *xyz_h;
  float3 ll, ur;
  unsigned int x,y,z;
  unsigned int n_part = dd.get_n_part();
  uint3 n_cells = dd.get_n_cells();
  unsigned int total_cells = n_cells.x * n_cells.y * n_cells.z;

  cells_h = (uint2 *)malloc(dd.total_cells*sizeof(uint2));
  xyz_h = (float3 *)malloc(n_part*sizeof(float3));

  hipMemcpy(cells_h, dd.cells, total_cells*sizeof(uint2), hipMemcpyDeviceToHost);
  hipMemcpy(xyz_h, dd.xyz_sorted, n_part*sizeof(float3), hipMemcpyDeviceToHost);  

  int *part_map = (int *)malloc(n_part*sizeof(int));
  memset(part_map, 0, n_part*sizeof(int));
  
  for(int i = 0; i < total_cells; i++) {    
    /* 3d index of the cell */
    z = i % n_cells.z;
    y = ((i - z)/n_cells.z) % n_cells.y;
    x = (i - z - n_cells.z*y) / (n_cells.z*n_cells.y);
    /* Cell boundaries */
    ll.x = x / dd.hi.x;
    ll.y = y / dd.hi.y;
    ll.z = z / dd.hi.z;
    ur.x = ll.x + 1./dd.hi.x;
    ur.y = ll.y + 1./dd.hi.y;
    ur.z = ll.z + 1./dd.hi.z;

    if(cells_h[i].x == CELL_EMPTY)
      continue;
    for(unsigned int it = cells_h[i].x; it != cells_h[i].y; it++) {      
      part_map[it]++;
      /* Check if particle blongs in this cell */
      if( ( ( xyz_h[it].x < ll.x ) || ( xyz_h[it].y < ll.y ) || ( xyz_h[it].z < ll.z ) ) ||
	  ( ( xyz_h[it].x >= ur.x ) || ( xyz_h[it].y >= ur.y ) || ( xyz_h[it].z >= ur.z ) ) ) {
	#ifdef GPU_DD_DEBUG
	printf("domain_decomposition_gpu: particle %d is in wrong cell.\n", it);
	#endif
	return false;
      }
    }
  }

  /* Check if all particles are in the dd */
  for(int i = 0; i < n_part; i++) {    
    if(part_map[i] != 1) {
      #ifdef GPU_DD_DEBUG
      printf("domain_decomposition_gpu: particle %d is missing or overcounted.\n", i);
      #endif
      return false;
    }
  }

  return true;
}

static bool test_neighbor_count() {
  float3 box;
  float cutoff;
  unsigned int n_part;
  
  box.x = 10;
  box.y = 20;
  box.z = 30;
  cutoff = 1.5;

  float3 parts[11] = { {0.0f, 00.f, 0.0f }, // 1 
		       {9.9f, 0.0f, 0.0f }, // 1
		       {6.0f, 5.0f, 5.0f }, // 6
		       {9.9f, 9.9f, 0.0f }, // 1
		       {5.0f, 5.0f, 5.0f }, // 7
		       {5.0f, 4.0f, 5.0f }, // 6
		       {0.0f, 9.9f, 0.0f }, // 1 
		       {5.0f, 5.0f, 4.0f }, // 6 
		       {5.0f, 6.0f, 5.0f }, // 6
		       {5.0f, 5.0f, 6.0f }, // 6
		       {4.0f, 5.0f, 5.0f }  // 6 
  };

unsigned int correct_count[11] = { 1, 1, 6, 1, 7, 6, 1, 6, 6, 6, 6 };

  n_part = sizeof(parts)/sizeof(float3);

  float3 *parts_d = 0;
  cuda_safe_mem(hipMalloc((void **)&parts_d, sizeof(parts)));
  cuda_safe_mem(hipMemcpy(parts_d, parts, sizeof(parts), hipMemcpyHostToDevice));
  
  DomainDecompositionGpu dd(box, n_part, cutoff);

  /* Check n_cells from box and cutoff */
  uint3 n_cells = dd.get_n_cells();
  if((n_cells.x != 6) || (n_cells.y != 13) || (n_cells.z != 20)) {
    return false;
  }

  /* Sort particles into cells */
  dd.build(parts_d);

  #ifdef GPU_DD_DEBUG
  print_dd(dd);
  #endif

  /* Check result */
  if(!test_decomposition(dd))
    return false;

  unsigned int *neighbors_dd_d, *neighbors_n2_d;
  unsigned int neighbors_dd[sizeof(parts)/sizeof(float3)];
  unsigned int neighbors_n2[sizeof(parts)/sizeof(float3)];

  cuda_safe_mem(hipMalloc((void **)&neighbors_dd_d, n_part * sizeof(unsigned int)));
  cuda_safe_mem(hipMalloc((void **)&neighbors_n2_d, n_part * sizeof(unsigned int)));

  dim3 block_dd(n_cells.x,n_cells.y,n_cells.z/2), grid_dd(1,1,n_cells.z/2);
  dim3 block_n2(n_part,1,1), grid_n2(1,1,1);
  float cutoff2 = cutoff*cutoff;

  /* Count using domain decomposition and sorted position array */
  KERNELCALL(nearestNeighbors, grid_dd, block_dd, (n_part, dd.get_n_cells(), dd.get_cells(), dd.get_xyz_sorted(), neighbors_dd_d,cutoff2));
  /* Count directly, also using sorted position array to get same particle order. */
  KERNELCALL(nearestNeighbors_n2, grid_n2, block_n2, (n_part, parts_d, neighbors_n2_d, cutoff2));

  cuda_safe_mem(hipMemcpy(neighbors_n2, neighbors_n2_d, n_part * sizeof(unsigned int), hipMemcpyDeviceToHost));

  dd.unsort<unsigned int>(neighbors_dd_d, neighbors_n2_d);
  cuda_safe_mem(hipMemcpy(neighbors_dd, neighbors_n2_d, n_part * sizeof(unsigned int), hipMemcpyDeviceToHost));
  
  for(int i = 0; i < n_part; i++) {
    if( (neighbors_dd[i] != correct_count[i]) || (neighbors_n2[i] != correct_count[i])) {
      #ifdef GPU_DD_DEBUG
      printf("%d %d %d\n", i, neighbors_dd[i], neighbors_n2[i]);
      #endif
      return false;
    }
  }
  cuda_safe_mem(hipFree(neighbors_dd_d));
  cuda_safe_mem(hipFree(neighbors_n2_d));
  cuda_safe_mem(hipFree(parts_d));

  return true;
}

/* Count neighbors wihtin rcut of each particle _without_ periodic boundaries using a domain decomposition */

__global__ static void nearestNeighbors(unsigned int n_part, uint3 n_cells, const uint2 *cells, const float3 *xyz, unsigned int *neighbors, float rcut2) {
  unsigned int xindex = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int yindex = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int zindex = blockDim.z * blockIdx.z + threadIdx.z;
  int n,m,l;
  unsigned int cellhash = n_cells.y * n_cells.z * xindex + n_cells.z * yindex + zindex;
  int hash;
  unsigned int m_neighbors = 0;

  if( (xindex >= n_cells.x) || (yindex >= n_cells.y) || (zindex >= n_cells.z))
    return;

  for(unsigned int id = cells[cellhash].x; id != cells[cellhash].y; id++) {
    m_neighbors = 0;
    float3 parti = xyz[id];
    for(int i = -1; i <= 1; i++) {
      n = xindex + i;
      if((n < 0) || (n >= n_cells.x))
	continue;
      for(int j = -1; j <= 1; j++) {
	m = yindex + j;
	if((m < 0) || (m >= n_cells.y))
	  continue;
	for(int k = -1; k <= 1; k++) {
	  l = zindex + k;
	  if((l < 0) || (l >= n_cells.z))
	    continue;

	  hash = n_cells.y * n_cells.z * n + n_cells.z * m + l;
	  if( (cells[hash].x == CELL_EMPTY))
	    continue;
	  for(unsigned int jd = cells[hash].x; jd != cells[hash].y; jd++) {
	    if(dist2(parti, xyz[jd]) <= rcut2) {
	      m_neighbors++;
	    }
	  }
	}
      }
    }
    neighbors[id] = m_neighbors;
  }
}

/* Count neighbors wihtin rcut2 of each particle _without_ periodic boundaries using a n2 loop */

__global__ static void nearestNeighbors_n2(unsigned int n_part, const float3 *xyz, unsigned int *neighbors, float rcut2) {
  unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int m_neighbors = 0;

  if(id >= n_part)
    return;

  float3 part = xyz[id]; 

  for(unsigned int i = 0; i < n_part; i++) {
    if(dist2(part, xyz[i]) <= rcut2) {
      m_neighbors++;
    }    
  }
  neighbors[id] = m_neighbors;
}



void print_dd(DomainDecompositionGpu &dd) {
  puts("print_dd()");
  uint2 *cells_h;
  float3 *xyz_h;
  unsigned int count = 0;

  cells_h = (uint2 *)malloc(dd.total_cells*sizeof(uint2));
  xyz_h = (float3 *)malloc(dd.n_part*sizeof(float3));

  hipMemcpy(cells_h, dd.cells, dd.total_cells*sizeof(uint2), hipMemcpyDeviceToHost);
  hipMemcpy(xyz_h, dd.xyz_sorted, dd.n_part*sizeof(float3), hipMemcpyDeviceToHost);  

  printf("dd { n_part = %d, n_cells = ( %d %d %d ), hi = ( %f %f %f ) }\n", dd.n_part, dd.n_cells.x, dd.n_cells.y, dd.n_cells.z
  	 , dd.hi.x, dd.hi.y, dd.hi.z);

  bool *part_map = (bool *)malloc(dd.n_part*sizeof(bool));
  for(int i = 0; i < dd.n_part; i++)
    part_map[i] = false;
  
  for(int i = 0; i < dd.total_cells; i++) {
    if(cells_h[i].x == CELL_EMPTY)
      continue;
    unsigned int x,y,z;
    z = i % dd.n_cells.z;
    y = ((i - z)/dd.n_cells.z) % dd.n_cells.y;
    x = (i - z - dd.n_cells.z*y) / (dd.n_cells.z*dd.n_cells.y);
    printf("cell %d, pos (%d %d %d), center (%f %f %f)\n", i, x, y, z, x/dd.hi.x, y/dd.hi.y, z/dd.hi.z);
    for(unsigned int it = cells_h[i].x; it != cells_h[i].y; it++) {
      part_map[it] = true;
      printf("\t%d: pos = (%f %f %f), dist = (%f %f %f), dist/h = (%f %f %f)\n", it,xyz_h[it].x,xyz_h[it].y,xyz_h[it].z,
  	     xyz_h[it].x-x/dd.hi.x, xyz_h[it].y-y/dd.hi.y, xyz_h[it].z-z/dd.hi.z,
  	     dd.hi.x*(xyz_h[it].x-x/dd.hi.x), dd.hi.y*(xyz_h[it].y-y/dd.hi.y), dd.hi.z*(xyz_h[it].z-z/dd.hi.z) );
      count++;
    }
  }
  printf("%d particles in dd.\n", count);
  for(int i = 0; i < dd.n_part; i++)
    if(!part_map[i])
      printf("particle %d is missing.\n",i);
}

